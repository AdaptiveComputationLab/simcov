#include "hip/hip_runtime.h"
#include "simcovcuda_driver.hpp"
#include "simcovcuda.hpp"

#define RNG_t hiprandState_t

/**
 * Cuda error checking
*/
inline void lastError(const char* kernel){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("#!!!CUDA Error in %s: %s\n",kernel, hipGetErrorString(err));
    }
}

/**
 * Device Functions
*/
__device__ bool inline d_isActive(Voxel* voxels, int i){
    if(voxels[i].virions > 0.0) return true;
    if(voxels[i].inflammation > 0.0) return true;
    if(voxels[i].cellType == EpiType::INCUBATING) return true;
    if(voxels[i].cellType == EpiType::EXPRESSING) return true;
    if(voxels[i].cellType == EpiType::APOPTOTIC) return true;
    if(voxels[i].voxelType == VoxelType::GHOST) return true;
    if(voxels[i].hasTCell) return true;
    return false;
}

__device__ float d_drawValue(RNG_t* state){
    float r = hiprand_uniform(state);
    return r;
}

__device__ uint d_drawUInt(RNG_t* state){
    unsigned int r = hiprand(state);
    return r;
}

__device__ int d_getBetween(RNG_t* state, int min, int max){
    if(min == max){
        return min;
    }
    uint r = d_drawUInt(state);
    return (int)((r)%((uint)max - (uint)min) + (uint)min);
}

__device__ unsigned int d_drawPoisson(RNG_t* state, float lambda){
    unsigned int r = hiprand_poisson(state, lambda);
    return r;
}

__device__ bool d_trialSuccess(RNG_t* state, float p){
    if(p > 1) return true;
    if(p < 0) return false;
    float roll = d_drawValue(state);
    if(roll < p) return true;
    return false;
}

__device__ void d_shuffle(RNG_t* state, int* a, int* b, int n){
    for(int i = 0; i < n; i++)
        b[i] = a[i];
    for(int i = n-1; i >= 1; i--){
        int j = d_getBetween(state, 0, i+1);
        int temp = b[j];
        b[j] = b[i];
        b[i] = temp;
    }
}

// Base functions for computing 3D and 1D coordinate transformations
__device__ inline void d_to3D(int i, Dimensions dims,
                int&dx, int&dy, int& dz){
    dx = i/(dims.z*dims.y);
    dy = (i % (dims.z*dims.y))/dims.z;
    dz = (i % (dims.z*dims.y))%dims.z;
}

__device__ inline int d_to1D(int x, int y, int z, Dimensions dims){
    return z + y*dims.z + x*(dims.y * dims.z);
}

__device__ inline bool d_inBoundaries(int x, int y, int z, Dimensions dims){
    if(x < 0 || x >= dims.x) return false;
    if(y < 0 || y >= dims.y) return false;
    if(z < 0 || z >= dims.z) return false;
    return true;
}

//functions for performing 3D to 1D coordinate transformations with
//tiles present
__device__ inline void d_to3D_tiled(int i,
                        Dimensions dims,
                        Dimensions tile,
                        Dimensions tileDims,
                        int& dx, int& dy, int& dz){
    
    //Determine the tile index
    int i_tile = i/tile.numPoints;

    //Compute the coordinates of the tile in 3D tile space
    int tx, ty, tz;
    d_to3D(i_tile, tileDims, tx, ty, tz);

    //get offset coords in voxel space
    //(i.e. the coords of the first voxel in this tile)
    int ox, oy, oz;
    ox = tx*tile.x;
    oy = ty*tile.y;
    oz = tz*tile.z;

    //Calculate the within tile coords
    int wi = i % tile.numPoints;
    int wx, wy, wz;
    d_to3D(wi, tile, wx, wy, wz);

    //result is offset + within tile coords
    dx = wx + ox;
    dy = wy + oy;
    dz = wz + oz;
}

__device__ inline int d_to1D_tiled(int x, int y, int z,
                    Dimensions dims,
                    Dimensions tile,
                    Dimensions tileDims){
    //get the tile coords in tile space
    int tx, ty, tz;
    tx = x/tile.x;
    ty = y/tile.y;
    tz = z/tile.z;
    //get the tile id
    int ti = d_to1D(tx, ty, tz, tileDims);

    //get the offset within the tile
    int ox, oy, oz;
    ox = x - tx*tile.x;
    oy = y - ty*tile.y;
    oz = z - tz*tile.z;
    int oi = d_to1D(ox, oy, oz, tile);

    //result is the total number of voxels up until this tile + the offest
    return ti*tile.numPoints + oi;
}

/**
 * Kernel definitions
**/
__global__ void k_initVoxels(Options opt, Comms comms, Dimensions dims, Dimensions tile, Dimensions tileDims,
                            hiprandState_t* deviceRNG,
                            Voxel* voxels){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = dims.numPoints;
    int step = blockDim.x*gridDim.x;

    //initialize rng on the device
    hiprand_init(opt.seed + (unsigned int)(2048*comms.rank), start, 0, &deviceRNG[start]);
    RNG_t localState = deviceRNG[start];

    for(int i = start; i < maximum; i += step){

        //voxels
        d_to3D_tiled(i, dims, tile, tileDims, voxels[i].x, voxels[i].y, voxels[i].z);
        Voxel* v = &voxels[i];

        if(v->x == 0 && comms.xPredecessor != -1) v->voxelType = VoxelType::GHOST;
        if(v->x == dims.x - 1 && comms.xSuccessor != -1) v->voxelType = VoxelType::GHOST;
        if(v->y == 0 && comms.yPredecessor != -1) v->voxelType = VoxelType::GHOST;
        if(v->y == dims.y - 1 && comms.ySuccessor != -1) v->voxelType = VoxelType::GHOST;
        if(v->z == 0 && comms.zPredecessor != -1) v->voxelType = VoxelType::GHOST;
        if(v->z == dims.z - 1 && comms.zSuccessor != -1) v->voxelType = VoxelType::GHOST;

        v->neighborhoodSize = 0;
        for(int ii = -1; ii <= 1; ii++){
            for(int jj = -1; jj <= 1; jj++){
                for(int kk = -1; kk <= 1; kk++){
                    int id = d_to1D_tiled(v->x + ii, v->y + jj, v->z + kk, dims, tile, tileDims);
                    if(d_inBoundaries(v->x + ii, v->y + jj, v->z + kk, dims)){
                        if(id == i){
                            v->myIdx = v->neighborhoodSize;
                        }
                        v->neighborhood[v->neighborhoodSize] = id;
                        v->neighborhoodSize++;
                    }
                }
            }
        }

        //epicells
        v->cellType = EpiType::HEALTHY;
        v->incubationTimeSteps = d_drawPoisson(&localState, opt.incubationPeriod);
        v->expressingTimeSteps = d_drawPoisson(&localState, opt.expressingPeriod);
        v->apoptoticTimeSteps = d_drawPoisson(&localState, opt.apoptosisPeriod);

        //concentration data
        v->virions = 0.0f;
        v->nbVirions = 0.0f;
        v->inflammation = 0.0f;
        v->nbInflammation = 0.0f;

        //set tcell data
        v->hasTCell = false;
        v->flip = false;
        v->fx = 0;
        v->fy = 0;
        v->fz = 0;
        v->tissueTimeSteps = -1;
        v->bindingPeriod = -1;
        v->bindProb = 2.0f;
        v->tieBreakValue = -1;
        v->winnerID = -1;
        v->winnerValue = -1;
    }
    deviceRNG[start] = localState;
}

__global__ void k_resetActives(Dimensions tileDims, int* tileMask, int* newTileMask){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = tileDims.numPoints;
    int step = blockDim.x*gridDim.x;

    for(int i = start; i < maximum; i+=step){
        newTileMask[i] = -1;
        tileMask[i] = -1;
    }
}

__global__ void k_checkActives(Dimensions dims, Dimensions tile, Dimensions tileDims,
                            Voxel* voxels,
                            int* tileMask, int* newTileMask, int* numActiveTiles){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = dims.numPoints;
    int step = blockDim.x*gridDim.x; 

    for(int i = start; i < maximum; i+=step){
        if(d_isActive(voxels, i)){
            //activate this tile
            int tileID = i/tile.numPoints;
            //must be atomicCAS if threaded
            atomicCAS(&(newTileMask[tileID]), -1, tileID);
            // activate neighbors
            // get x,y,z
            int x,y,z;
            d_to3D(tileID, tileDims, x, y, z);

            for(int dx = -1; dx <= 1; dx++){
                for(int dy = -1; dy <= 1; dy++){
                    for(int dz = -1; dz <= 1; dz++){
                        int nx = x + dx;
                        int ny = y + dy;
                        int nz = z + dz;
                        if(d_inBoundaries(nx, ny, nz, tileDims)){
                            int ni = d_to1D(nx, ny, nz, tileDims);
                            if(ni != i){
                                //must be atomicCAS if threaded
                                atomicCAS(&(newTileMask[ni]), -1, ni);
                            }
                        }
                    }
                }
            }
        }
    }
}

__global__ void k_sortActives(Dimensions tileDims, int* tileMask, int* newTileMask, int* numActiveTiles){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = tileDims.numPoints;
    int step = blockDim.x*gridDim.x; 

    for(int i = start; i < maximum; i+=step){

        //check if I'm active
        if(newTileMask[i] != -1){
            int prev = atomicAdd(numActiveTiles, 1);
            tileMask[prev] = i;
        }
    }
}

__global__ void k_setSingleValue(int* data, int i, int value){
    data[i] = value;
}

__global__ void k_accumulate(Options opt, Dimensions dims, Dimensions tile,
                            Voxel* voxels,
                            int* tileMask, int* numActiveTiles){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileID = tileMask[idx / (tile.numPoints)];
        if(tileID == -1) continue;
        int i = tileID*tile.numPoints + idx%tile.numPoints;

        // if(voxels[i].voxelType == VoxelType::GHOST) continue;

        Voxel* v = &voxels[i];
        v->nbVirions = 0.0f;
        v->nbInflammation = 0.0f;
        for(int j = 0; j < v->neighborhoodSize; j++){
            int nbID = v->neighborhood[j];
            if(nbID != i){
                v->nbVirions += voxels[nbID].virions;
                v->nbInflammation += voxels[nbID].inflammation;
            }
        }
    }
}

__global__ void  k_spread(Options opt, Dimensions dims, Dimensions tile,
                            Voxel* voxels,
                            int* tileMask, int* numActiveTiles){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileID = tileMask[idx / (tile.numPoints)];
        if(tileID == -1) continue;
        int i = tileID*tile.numPoints + idx%tile.numPoints;

        // if(voxels[i].voxelType == VoxelType::GHOST) continue;

        Voxel* v = &voxels[i];

        //virion
        float virionsDiffused = v->virions*opt.virionDiffusion;
        float virionsLeft = v->virions - virionsDiffused;
        float avgNBVirions = (virionsDiffused + v->nbVirions*opt.virionDiffusion)/(v->neighborhoodSize);

        v->virions = virionsLeft + avgNBVirions;

        v->virions = (1.0 - opt.virionClearance)*v->virions;
        if(v->virions < opt.minVirions) v->virions = 0.0;
        v->nbVirions = 0.0f;

        //inflammation
        float inflammationDiffused = v->inflammation*opt.inflammationDiffusion;
        float inflammationLeft = v->inflammation - inflammationDiffused;
        float avgNBInflammation = (inflammationDiffused + v->nbInflammation*opt.inflammationDiffusion)/(v->neighborhoodSize);

        v->inflammation = inflammationLeft + avgNBInflammation;
        v->inflammation = (1.0 - opt.inflammationDecay)*v->inflammation;
        if(v->inflammation < opt.minInflammation) v->inflammation = 0.0f;
        v->nbInflammation = 0.0f;

    }
}

__global__ void k_updateEpiCells(Options opt, Dimensions dims, Dimensions tile,
                                Voxel* voxels, hiprandState_t* deviceRNG,
                                int* tileMask, int* numActiveTiles){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;
    RNG_t localState;

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileID = tileMask[idx / (tile.numPoints)];
        if(tileID == -1) continue;
        int i = tileID*tile.numPoints + idx%tile.numPoints;

        localState = deviceRNG[i % (opt.threadsPerBlock*opt.blocks)];

        if(voxels[i].voxelType == VoxelType::GHOST) continue;

        bool produceVirions = false;
        switch(voxels[i].cellType) {
            case EpiType::HEALTHY:
                if(voxels[i].virions > 0) {
                    if(d_trialSuccess(&localState, voxels[i].virions*opt.infectivity)) {
                        voxels[i].cellType = EpiType::INCUBATING;
                    }
                }
                break;
            case EpiType::INCUBATING:
                voxels[i].incubationTimeSteps--;
                if(voxels[i].incubationTimeSteps <= 0) {
                    voxels[i].cellType = EpiType::EXPRESSING;
                }
                break;
            case EpiType::EXPRESSING:
                voxels[i].expressingTimeSteps--;
                if(voxels[i].expressingTimeSteps <= 0) {
                    voxels[i].cellType = EpiType::DEAD;
                } else {
                    produceVirions = true;
                }
                break;
            case EpiType::APOPTOTIC:
                voxels[i].apoptoticTimeSteps--;
                if(voxels[i].apoptoticTimeSteps <= 0) {
                    voxels[i].cellType = EpiType::DEAD;
                } else if (voxels[i].incubationTimeSteps<=0) {
                    produceVirions = true;
                }
                break;
            default: break;
        }
        if(produceVirions) {
            voxels[i].virions = opt.virionProduction + voxels[i].virions;
            voxels[i].inflammation = opt.inflammationProduction + voxels[i].inflammation;
            if(voxels[i].inflammation > 1.0) {
                voxels[i].inflammation = 1.0;
            }
        }
        deviceRNG[i % (opt.threadsPerBlock*opt.blocks)] = localState;
    }
    
}

__global__ void k_reduce(Dimensions dims,
                        Voxel* voxels,
                        Globals* blockGlobals,
                        int offset, int threads){
    extern __shared__ float s[];
    float *sharedVirions = s;
    float *sharedInflammation = (float*)&sharedVirions[threads];
    int *sharedTCells = (int*)&sharedInflammation[threads];
    int *sharedHealthy = (int*)&sharedTCells[threads];
    int *sharedIncubating = (int*)&sharedHealthy[threads];
    int *sharedExpressing = (int*)&sharedIncubating[threads];
    int *sharedApoptotic = (int*)&sharedExpressing[threads];
    int *sharedDead = (int*)&sharedApoptotic[threads];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x + offset;

    sharedVirions[tid] = 0;
    sharedInflammation[tid] = 0;
    sharedTCells[tid] = 0;
    sharedHealthy[tid] = 0;
    sharedIncubating[tid] = 0;
    sharedExpressing[tid] = 0;
    sharedApoptotic[tid] = 0;
    sharedDead[tid] = 0;

    if(i < dims.numPoints && voxels[i].voxelType != VoxelType::GHOST){
        sharedVirions[tid] = voxels[i].virions;
        sharedInflammation[tid] = voxels[i].inflammation;
        if(voxels[i].hasTCell) sharedTCells[tid] = 1;
        switch(voxels[i].cellType){
            case EpiType::HEALTHY:
                sharedHealthy[tid] = 1;
                break;
            case EpiType::INCUBATING:
                sharedIncubating[tid] = 1;
                break;
            case EpiType::EXPRESSING:
                sharedExpressing[tid] = 1;
                break;
            case EpiType::APOPTOTIC:
                sharedApoptotic[tid] = 1;
                break;
            case EpiType::DEAD:
                sharedDead[tid] = 1;
                break;
            default:
                break;
        }
    }
    __syncthreads();

    for(unsigned int s=blockDim.x/2; s > 0; s >>=1){
        int index = tid;
        if(index < s){
            sharedVirions[index] += sharedVirions[index + s];       
            sharedInflammation[index] += sharedInflammation[index + s];
            sharedTCells[index] += sharedTCells[index + s];
            sharedHealthy[index] += sharedHealthy[index + s];       
            sharedIncubating[index] += sharedIncubating[index + s];       
            sharedExpressing[index] += sharedExpressing[index + s];       
            sharedApoptotic[index] += sharedApoptotic[index + s];       
            sharedDead[index] += sharedDead[index + s];       
        }
        __syncthreads();
    }

    //write back to global memory
    if (tid == 0) {
        blockGlobals[blockIdx.x].totalVirions = sharedVirions[0];
        blockGlobals[blockIdx.x].totalInflammation = sharedInflammation[0];
        blockGlobals[blockIdx.x].totalTCells = sharedTCells[0];
        blockGlobals[blockIdx.x].totalHealthy = sharedHealthy[0];
        blockGlobals[blockIdx.x].totalIncubating = sharedIncubating[0];
        blockGlobals[blockIdx.x].totalExpressing = sharedExpressing[0];
        blockGlobals[blockIdx.x].totalApoptotic = sharedApoptotic[0];
        blockGlobals[blockIdx.x].totalDead = sharedDead[0];
    }
}

__global__ void k_reduceBlocks(Globals* blockGlobals, Globals* globals){
    int i = threadIdx.x;
    atomicAdd(&(globals->totalVirions), blockGlobals[i].totalVirions);
    atomicAdd(&(globals->totalInflammation), blockGlobals[i].totalInflammation);
    atomicAdd(&(globals->totalHealthy), blockGlobals[i].totalHealthy);
    atomicAdd(&(globals->totalIncubating), blockGlobals[i].totalIncubating);
    atomicAdd(&(globals->totalExpressing), blockGlobals[i].totalExpressing);
    atomicAdd(&(globals->totalApoptotic), blockGlobals[i].totalApoptotic);
    atomicAdd(&(globals->totalDead), blockGlobals[i].totalDead);
    atomicAdd(&(globals->totalTCells), blockGlobals[i].totalTCells);
}

__global__ void k_resetGlobals(Globals* globals){
    globals->totalVirions = 0.0;
    globals->totalInflammation = 0.0;
    globals->totalHealthy = 0.0;
    globals->totalIncubating = 0.0;
    globals->totalExpressing = 0.0;
    globals->totalApoptotic = 0.0;
    globals->totalDead = 0.0;
    globals->totalTCells = 0.0;
}

__global__ void k_initializeInfection(float count, int x, int y, int z, Dimensions dims, Dimensions tile, Dimensions tileDims, Voxel* voxels, Comms comms){
    //check if I own this tile
    int minX, maxX;
    int minY, maxY;
    int minZ, maxZ;
    minX = comms.rankX*comms.rankDimX;
    maxX = minX + comms.rankDimX;
    minY = comms.rankY*comms.rankDimY;
    maxY = minY + comms.rankDimY;
    minZ = comms.rankZ*comms.rankDimZ;
    maxZ = minZ + comms.rankDimZ;

    if(x < minX || x >= maxX) return;
    if(y < minY || y >= maxY) return;
    if(z < minZ || z >= maxZ) return;

    int ox = x - minX;
    int oy = y - minY;
    int oz = z - minZ;
    if(comms.xPredecessor != -1) ox++;
    if(comms.yPredecessor != -1) oy++;
    if(comms.zPredecessor != -1) oz++;

    int i = d_to1D_tiled(ox, oy, oz, dims, tile, tileDims);
    voxels[i].virions = count;
    printf("### Rank %d initialized infection %f at %d = (%d,%d,%d) locally at (%d,%d,%d)\n", comms.rank, count, i, x,y,z, ox,oy,oz);
    voxels[i].cellType = EpiType::INCUBATING;
}

__global__ void k_ageTCells(Dimensions dims, Dimensions tile,
                            Voxel* voxels,
                            int* tileMask, int* numActiveTiles){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileid = tileMask[idx / (tile.numPoints)];
        if(tileid == -1) continue;
        int i = tileid*tile.numPoints + idx%tile.numPoints;

        if(voxels[i].voxelType == VoxelType::GHOST) continue;

        voxels[i].tissueTimeSteps--;
        if(voxels[i].tissueTimeSteps <= 0) {
            //kill the tcell
            voxels[i].tissueTimeSteps = -1;
            voxels[i].bindingPeriod = -1;
            voxels[i].hasTCell = false;
            voxels[i].flip = false;
        }
    }
}

__global__ void k_setupBind(Options opt, Dimensions dims, Dimensions tile, Dimensions tileDims,
                            Voxel* voxels, hiprandState_t* deviceRNG,
                            int* tileMask, int* numActiveTiles){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;
    int stateIDX = start%(opt.threadsPerBlock*opt.blocks);
    RNG_t localState = deviceRNG[stateIDX];

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileID = tileMask[idx / (tile.numPoints)];
        if(tileID == -1) continue;
        int i = tileID*tile.numPoints + idx%tile.numPoints;

        if(voxels[i].voxelType == VoxelType::GHOST) continue;

        Voxel* v = &voxels[i];
        v->bindProb = 2.0f; //reset

        if(v->bindingPeriod != -1){
            v->bindingPeriod--;
            if(v->bindingPeriod <= 0){
                v->bindingPeriod = -1;
                v->fx = 0;
                v->fy = 0;
                v->fz = 0;
                v->bindProb = 2.0f; //ensures won't bind this turn
            }
            continue;
        }
        if(!v->hasTCell) continue;

        v->fx = 0;
        v->fy = 0;
        v->fz = 0;
        bool foundTarget = false;
        int dx = 0, dy = 0, dz = 0;

        //shuffle neighborhood
        int* shuffledNeighbors = new int[v->neighborhoodSize];
        d_shuffle(&localState, v->neighborhood, shuffledNeighbors, v->neighborhoodSize);

        for(int ii = 0; ii < v->neighborhoodSize; ii++){
            int ni = shuffledNeighbors[ii];
            Voxel vn = voxels[ni];
            v->bindProb = d_drawValue(&localState);
            dx = vn.x - v->x;
            dy = vn.y - v->y;
            dz = vn.z - v->z;

            if(voxels[ni].cellType == EpiType::HEALTHY || voxels[ni].cellType == EpiType::DEAD){
                continue;
            }

            double scaling = 1.0 - (double)voxels[ni].incubationTimeSteps/opt.incubationPeriod;

            if(voxels[ni].cellType == EpiType::EXPRESSING || voxels[ni].cellType == EpiType::APOPTOTIC){
                scaling = opt.maxBindingProb;
            }

            if(scaling < 0) scaling = 0;
            double prob = opt.maxBindingProb*scaling;
            if(prob >= opt.maxBindingProb) prob = opt.maxBindingProb;

            if(v->bindProb < prob){
                foundTarget = true;
                v->bindProb = -1.0; //i will bind
                break;
            }
        }
        delete [] shuffledNeighbors;

        if(foundTarget){
            v->fx = dx;
            v->fy = dy;
            v->fz = dz;
        }

        if(dims.x == 1) v->fx = 0;
        if(dims.y == 1) v->fy = 0;
        if(dims.z == 1) v->fz = 0;
    }
    deviceRNG[stateIDX] = localState;
}

__device__ EpiType atomicLoad(const EpiType* addr){
    const volatile EpiType *vaddr = addr;
    __threadfence();
    const EpiType value = *vaddr;
    __threadfence();
    return value;
}

__device__ void atomicStore(EpiType* addr, EpiType value){
    volatile EpiType *vaddr = addr;
    __threadfence();
    *vaddr = value;
}

__global__ void k_executeBind(Options opt, Dimensions dims, Dimensions tile, Dimensions tileDims,
                            Voxel* voxels,
                            int* tileMask, int* numActiveTiles){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileID = tileMask[idx / (tile.numPoints)];
        if(tileID == -1) continue;
        int i = tileID*tile.numPoints + idx%tile.numPoints;
        Voxel* v = &voxels[i];
        int lx, ly, lz;
        d_to3D_tiled(i, dims, tile, tileDims, lx, ly, lz);
        if(v->bindingPeriod == -1 && v->hasTCell){
            if(d_inBoundaries(lx + v->fx, ly + v->fy, lz + v->fz, dims)){
                int id = d_to1D_tiled(lx + v->fx, ly + v->fy, lz + v->fz, dims, tile, tileDims);
                if(v->bindProb < 0.0){
                    v->bindingPeriod = opt.tcellBindingPeriod;
                    atomicStore(&voxels[id].cellType, EpiType::APOPTOTIC);
                }
            }
            v->bindProb = 0.0;

            //dont move if bound
            if(v->bindingPeriod != -1){
                v->fx = 0;
                v->fy = 0;
                v->fz = 0;
            }

        }
    }
}

__global__ void k_setupMove(Dimensions dims, Dimensions tile,
                            Voxel* voxels, hiprandState_t* deviceRNG,
                            int* tileMask, int* numActiveTiles){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;

    RNG_t localState = deviceRNG[start];

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileID = tileMask[idx / (tile.numPoints)];
        if(tileID == -1) continue;
        int i = tileID*tile.numPoints + idx%tile.numPoints;

        if(voxels[i].voxelType == VoxelType::GHOST) continue;

        Voxel* v = &voxels[i];

        if(v->hasTCell && v->bindingPeriod == -1){
            //choose a random direction to face
            int nbIDX = d_getBetween(&localState, 0, v->neighborhoodSize-1);
            if(nbIDX >= v->myIdx) nbIDX += 1;
            int nbID = v->neighborhood[nbIDX];
            v->fx = voxels[nbID].x - v->x;
            v->fy = voxels[nbID].y - v->y;
            v->fz = voxels[nbID].z - v->z;

            v->tieBreakValue = d_getBetween(&localState, 0, 1000000);
            if(v->tieBreakValue > voxels[nbID].winnerValue){
                atomicCAS(&voxels[nbID].winnerValue,
                        voxels[nbID].winnerValue,
                        v->tieBreakValue);
            } else {
                v->fx = 0;
                v->fy = 0;
                v->fz = 0;
                v->tieBreakValue = -1;
            }
        }
    }
    deviceRNG[start] = localState;
}

__global__ void k_declareWinners(Dimensions dims, Dimensions tile,
                                Voxel* voxels,
                                int* tileMask, int* numActiveTiles){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileID = tileMask[idx / (tile.numPoints)];
        if(tileID == -1) continue;
        int i = tileID*tile.numPoints + idx%tile.numPoints;

        Voxel* v = &voxels[i];
        if(v->tieBreakValue != -1){
            for(int ii = 0; ii < v->neighborhoodSize; ii++){
                int nbID = v->neighborhood[ii];
                if(voxels[nbID].winnerValue == v->tieBreakValue){
                    if(!voxels[nbID].hasTCell){
                        voxels[nbID].winnerID = i;
                        v->flip = true;
                        voxels[nbID].flip = true;
                        voxels[nbID].bindingPeriod = v->bindingPeriod;
                        voxels[nbID].tissueTimeSteps = v->tissueTimeSteps;
                        voxels[nbID].bindProb = v->bindProb;
                    }
                    break;
                }
            }
        }

    }
}

__global__ void k_flipTCells(Dimensions dims, Dimensions tile,
                            Voxel* voxels,
                            int* tileMask, int* numActiveTiles){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = (*numActiveTiles)*tile.numPoints;
    int step = blockDim.x*gridDim.x;

    for(int idx = start; idx < maximum; idx += step){
        //stay within sim bounds even if tiles extend beyond
        if(idx >= dims.numPoints || idx < 0) continue;

        //figure out index into data array
        int tileID = tileMask[idx / (tile.numPoints)];
        if(tileID == -1) continue;
        int i = tileID*tile.numPoints + idx%tile.numPoints;

        Voxel* v = &voxels[i];

        if(v->flip){
            if(v->hasTCell){
                //unset tcell values
                v->bindingPeriod = -1;
                v->tissueTimeSteps = -1;
                v->bindProb = 2.0f;
                v->hasTCell = false;
            } else {
                v->hasTCell = true;
            }
        }
        //reset random walk values
        v->tieBreakValue = -1;
        v->winnerValue = -1;
        v->winnerID = -1;
        v->flip = false;
    }
}

__global__ void k_spawnTCells(int numXTravasing,
                            Options opt, Comms comms, Dimensions dims, Dimensions tile, Dimensions tileDims,
                            Voxel* voxels, Globals* globals, hiprandState_t* deviceRNG){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = numXTravasing;
    int step = blockDim.x*gridDim.x;

    int stateIDX = start%(opt.threadsPerBlock*opt.blocks);
    RNG_t localState = deviceRNG[stateIDX];

    for(int i = start; i < maximum; i += step) {

        int minX = 0, maxX = dims.x;
        int minY = 0, maxY = dims.y;
        int minZ = 0, maxZ = dims.z;

        //deal with ghost voxels
        if(comms.xPredecessor != -1) minX += 1;
        if(comms.xSuccessor != -1) maxX -= 1;
        if(comms.yPredecessor != -1) minY += 1;
        if(comms.ySuccessor != -1) maxY -= 1;
        if(comms.zPredecessor != -1) minZ += 1;
        if(comms.zSuccessor != -1) maxZ -= 1;

        int lx = d_getBetween(&localState, minX, maxX);
        int ly = d_getBetween(&localState, minY, maxY);
        int lz = d_getBetween(&localState, minZ, maxZ);

        int id = d_to1D_tiled(lx,ly,lz,dims, tile, tileDims);
        int lifeTime = d_drawPoisson(&localState, opt.tcellTissuePeriod);

        if(voxels[id].inflammation < opt.minInflammation) continue;
        int prev_id = atomicCAS(&voxels[id].tissueTimeSteps, -1, lifeTime);
        if(prev_id != -1) continue;
        voxels[id].hasTCell = true;
        atomicAdd(&globals->numCirculatingTCells, (unsigned long long int)(-1));
    }
    deviceRNG[stateIDX] = localState;
}

__global__ void k_packSlice(Options& opt,
                        Voxel* data, Voxel* packedBuffer,
                        Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms,
                        int sliceDim, int sliceIdx){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = 0;
    int step = blockDim.x*gridDim.x;

    //X
    if(sliceDim == 0){
        maximum = comms.xSliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = sliceIdx;
            y = i / dims.z;
            z = i % dims.z;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);

            packedBuffer[i] = data[idx];
        }
    }

    //Y
    if(sliceDim == 1){
        maximum = comms.ySliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = i % dims.x;
            y = sliceIdx;
            z = i / dims.x;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            packedBuffer[i] = data[idx];
        }
    }

    //Z
    if(sliceDim == 2){
        maximum = comms.zSliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = i / dims.y;
            y = i % dims.y;
            z = sliceIdx;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            packedBuffer[i] = data[idx];
        }
    }
    
}

__global__ void k_unpackSlice(Options& opt,
                        Voxel* data, Voxel* packedBuffer,
                        Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms,
                        int sliceDim, int sliceIdx){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = 0;
    int step = blockDim.x*gridDim.x;

    //X
    if(sliceDim == 0){
        maximum = comms.xSliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = sliceIdx;
            y = i / dims.z;
            z = i % dims.z;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            //preserve structure data
            int neighborhoodSize = data[idx].neighborhoodSize;
            int neighborhood[27] = {0};
            for(int ii = 0; ii < 27; ii++){
                neighborhood[ii] = data[idx].neighborhood[ii];
            }
            int myIdx = data[idx].myIdx;
            VoxelType voxelType = data[idx].voxelType;
            int oldx = data[idx].x;
            int oldy = data[idx].y;
            int oldz = data[idx].z;

            //copy
            data[idx] = packedBuffer[i];
            
            //reset structure data
            data[idx].neighborhoodSize = neighborhoodSize;
            for(int ii = 0; ii < 27; ii++){
                data[idx].neighborhood[ii] = neighborhood[ii];
            }
            data[idx].myIdx = myIdx;
            data[idx].voxelType = voxelType;
            data[idx].x = oldx;
            data[idx].y = oldy;
            data[idx].z = oldz;
        }
    }

    //Y
    if(sliceDim == 1){
        maximum = comms.ySliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = i % dims.x;
            y = sliceIdx;
            z = i / dims.x;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            //preserve structure data
            int neighborhoodSize = data[idx].neighborhoodSize;
            int neighborhood[27] = {0};
            for(int ii = 0; ii < 27; ii++){
                neighborhood[ii] = data[idx].neighborhood[ii];
            }
            int myIdx = data[idx].myIdx;
            VoxelType voxelType = data[idx].voxelType;
            int oldx = data[idx].x;
            int oldy = data[idx].y;
            int oldz = data[idx].z;

            //copy
            data[idx] = packedBuffer[i];
            
            //reset structure data
            data[idx].neighborhoodSize = neighborhoodSize;
            for(int ii = 0; ii < 27; ii++){
                data[idx].neighborhood[ii] = neighborhood[ii];
            }
            data[idx].myIdx = myIdx;
            data[idx].voxelType = voxelType;
            data[idx].x = oldx;
            data[idx].y = oldy;
            data[idx].z = oldz;
        }
    }

    //Z
    if(sliceDim == 2){
        maximum = comms.zSliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = i / dims.y;
            y = i % dims.y;
            z = sliceIdx;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
           //preserve structure data
            int neighborhoodSize = data[idx].neighborhoodSize;
            int neighborhood[27] = {0};
            for(int ii = 0; ii < 27; ii++){
                neighborhood[ii] = data[idx].neighborhood[ii];
            }
            int myIdx = data[idx].myIdx;
            VoxelType voxelType = data[idx].voxelType;
            int oldx = data[idx].x;
            int oldy = data[idx].y;
            int oldz = data[idx].z;

            //copy
            data[idx] = packedBuffer[i];
            
            //reset structure data
            data[idx].neighborhoodSize = neighborhoodSize;
            for(int ii = 0; ii < 27; ii++){
                data[idx].neighborhood[ii] = neighborhood[ii];
            }
            data[idx].myIdx = myIdx;
            data[idx].voxelType = voxelType;
            data[idx].x = oldx;
            data[idx].y = oldy;
            data[idx].z = oldz;
        }
    }
    
}


__global__ void k_generateTCells(Options opt, Globals* globals){
    globals->numCirculatingTCells += (unsigned long long int)(opt.tcellGenerationRate);
}

/**
 * Kernel Wrappers
*/
void initVoxels(Options opt, Comms comms, Dimensions dims, Dimensions tile, Dimensions tileDims,
                hiprandState_t* deviceRNG, Voxel* voxels){
    k_initVoxels<<<opt.blocks, opt.threadsPerBlock>>>(opt, comms, dims, tile, tileDims,
                                                     deviceRNG, voxels);
    lastError("initVoxels");
    hipDeviceSynchronize();
}

void checkActiveTiles(Options opt, int timeStep,
                    Dimensions dims, Dimensions tile, Dimensions tileDims,
                    Voxel* voxels,
                    int* tileMask, int* newTileMask, int* numActiveTiles){
    if(timeStep % opt.checkActivesRate == 0){
        k_resetActives<<<opt.blocks, opt.threadsPerBlock>>>(tileDims, tileMask, newTileMask);
        lastError("resetActives");
        
        k_setSingleValue<<<1,1>>>(numActiveTiles, 0, 0); //Might be a smarter way to do this but it isn't very expensive. Faster than copying anyway
        lastError("setSingleValue");

        k_checkActives<<<opt.blocks, opt.threadsPerBlock>>>(dims, tile, tileDims,
                                                            voxels,
                                                            tileMask, newTileMask, numActiveTiles);
        lastError("checkActives");

        k_sortActives<<<opt.blocks, opt.threadsPerBlock>>>(tileDims, tileMask, newTileMask, numActiveTiles);
        lastError("sortActives");
    }
}

void updateConcentrations(Options opt, Dimensions dims, Dimensions tile,
                            Voxel* voxels,
                            int* tileMask, int* numActiveTiles){
    k_accumulate<<<opt.blocks, opt.threadsPerBlock>>>(opt, dims, tile,
                                                    voxels,
                                                    tileMask, numActiveTiles);
    lastError("update concentrations");

    k_spread<<<opt.blocks, opt.threadsPerBlock>>>(opt, dims, tile,
                                                voxels,
                                                tileMask, numActiveTiles);
    lastError("spread");
}
void updateEpithelialCells(Options opt, Dimensions dims, Dimensions tile,
                                Voxel* voxels, hiprandState_t* deviceRNG,
                                int* tileMask, int* numActiveTiles){
    k_updateEpiCells<<<opt.blocks, opt.threadsPerBlock>>>(opt, dims, tile,
                                voxels, deviceRNG,
                                tileMask, numActiveTiles);
    lastError("updateEpithelialCells");
}

void reduceGPU(Options opt,
            Dimensions dims, Voxel* voxels,
            Globals* blockGlobals, Globals* globals){
    int threads = opt.threadsPerBlock;
    int blocks = opt.blocks;
    int shmem_size = threads*(2*sizeof(float) + 6*sizeof(int));
    k_resetGlobals<<<1,1>>>(globals);
    for(int i = 0; i < dims.numPoints; i+=(threads*blocks))
    {
        k_reduce<<<blocks, threads, shmem_size>>>(dims, voxels, blockGlobals, i, threads);
        lastError("reduce");
        k_reduceBlocks<<<1, blocks>>>(blockGlobals, globals);
        lastError("reduceBlocks");
    }
    hipDeviceSynchronize();
}

void initializeInfection(float count, int x, int y, int z, Dimensions dims, Dimensions tile, Dimensions tileDims, Voxel* voxels, Comms comms){
    k_initializeInfection<<<1,1>>>(count, x, y, z, dims, tile, tileDims, voxels, comms);
    lastError("initializeInfection");
}

void ageTCells(Options opt, Dimensions dims, Dimensions tile,
                            Voxel* voxels,
                            int* tileMask, int* numActiveTiles){
    k_ageTCells<<<opt.blocks,opt.threadsPerBlock>>>(dims, tile, voxels, tileMask, numActiveTiles);
    lastError("ageTCells");
}

void setupBind(Options opt, Dimensions dims, Dimensions tile, Dimensions tileDims,
                            Voxel* voxels, hiprandState_t* deviceRNG,
                            int* tileMask, int* numActiveTiles){
    k_setupBind<<<opt.blocks, opt.threadsPerBlock>>>(opt, dims, tile, tileDims,
                                                    voxels, deviceRNG,
                                                    tileMask, numActiveTiles);
}

void executeBind(Options opt, Dimensions dims, Dimensions tile, Dimensions tileDims,
                            Voxel* voxels,
                            int* tileMask, int* numActiveTiles){
    k_executeBind<<<opt.blocks, opt.threadsPerBlock>>>(opt, dims, tile, tileDims,
                                                    voxels,
                                                    tileMask, numActiveTiles);
    lastError("executeBind");
}

void setupMove(Options opt, Dimensions dims, Dimensions tile,
                            Voxel* voxels, hiprandState_t* deviceRNG,
                            int* tileMask, int* numActiveTiles){
    k_setupMove<<<opt.blocks, opt.threadsPerBlock>>>(dims, tile, voxels, deviceRNG, tileMask, numActiveTiles);
    lastError("setupMove");
}

void declareWinners(Options opt, Dimensions dims, Dimensions tile,
                                Voxel* voxels,
                                int* tileMask, int* numActiveTiles){
    k_declareWinners<<<opt.blocks, opt.threadsPerBlock>>>(dims, tile, voxels, tileMask, numActiveTiles);
    lastError("declare winners");
}

void flipTCells(Options opt, Dimensions dims, Dimensions tile, Voxel* voxels,
                int* tileMask, int* numActiveTiles){
    k_flipTCells<<<opt.blocks, opt.threadsPerBlock>>>(dims, tile, voxels, tileMask, numActiveTiles);
    lastError("flipTCells");
}

void spawnTCells(int numXTravasing,
    Options opt, Comms comms, Dimensions dims, Dimensions tile, Dimensions tileDims,
    Voxel* voxels, Globals* globals, hiprandState_t* deviceRNG){
    k_spawnTCells<<<opt.blocks, opt.threadsPerBlock>>>(numXTravasing,
                                                        opt, comms, dims, tile, tileDims,
                                                        voxels, globals, deviceRNG);
}

void generateTCells(Options opt, Globals* globals){
    k_generateTCells<<<1, 1>>>(opt, globals);
    lastError("generateTCells");
}

void packSlice(Options& opt,
                        Voxel* data, Voxel* packedBuffer,
                        Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms,
                        int sliceDim, int sliceIdx){
    k_packSlice<<<opt.blocks, opt.threadsPerBlock>>>(opt, data, packedBuffer, dims, tile, tileDims, comms, sliceDim, sliceIdx);
    hipDeviceSynchronize();
}

void unpackSlice(Options& opt,
                        Voxel* data, Voxel* packedBuffer,
                        Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms,
                        int sliceDim, int sliceIdx){
    k_unpackSlice<<<opt.blocks, opt.threadsPerBlock>>>(opt, data, packedBuffer, dims, tile, tileDims, comms, sliceDim, sliceIdx);
    hipDeviceSynchronize();
}

//host rng functions
float drawValue(hiprandGenerator_t& gen){
    float r;
    hiprandGenerateUniform(gen, &r, 1);
    return r;
}

uint drawUInt(hiprandGenerator_t& gen){
    unsigned int r;
    hiprandGenerate(gen, &r, 1);
    return r;
}

int getBetween(hiprandGenerator_t& gen, int min, int max){
    if(min == max){
        return min;
    }
    uint r = drawUInt(gen);
    return (int)((r)%((uint)max - (uint)min) + (uint)min);
}

unsigned int drawPoisson(hiprandGenerator_t& gen, float lambda){
    unsigned int r;
    hiprandGeneratePoisson(gen, &r, 1, lambda);
    return r;
}

bool trialSuccess(hiprandGenerator_t& gen, float p){
    if(p > 1) return true;
    if(p < 0) return false;
    float roll = drawValue(gen);
    if(roll < p) return true;
    return false;
}

void setupHostRNG(hiprandGenerator_t& gen, Options& opt){
    hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, opt.seed);
}

__global__ void k_setCommTestData(Options opt, testData* data,
                                Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms){
    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = dims.numPoints;
    int step = blockDim.x*gridDim.x;

    for(int i = start; i < maximum; i += step){
        data[i].rank = comms.rank;
        d_to3D_tiled(i, dims, tile, tileDims, data[i].x, data[i].y, data[i].z);
    }
}

__global__ void k_packSliceTestData(Options& opt,
                        testData* data, testData* packedBuffer,
                        Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms,
                        int sliceDim, int sliceIdx){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = 0;
    int step = blockDim.x*gridDim.x;

    //X
    if(sliceDim == 0){
        maximum = comms.xSliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = sliceIdx;
            y = i / dims.z;
            z = i % dims.z;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            packedBuffer[i].rank = data[idx].rank;
        }
    }

    //Y
    if(sliceDim == 1){
        maximum = comms.ySliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = i % dims.x;
            y = sliceIdx;
            z = i / dims.x;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            packedBuffer[i].rank = data[idx].rank;
        }
    }

    //Z
    if(sliceDim == 2){
        maximum = comms.zSliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = i / dims.y;
            y = i % dims.y;
            z = sliceIdx;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            packedBuffer[i].rank = data[idx].rank;
        }
    }
    
}

__global__ void k_unpackSliceTestData(Options& opt,
                        testData* data, testData* packedBuffer,
                        Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms,
                        int sliceDim, int sliceIdx){

    int start = blockIdx.x*blockDim.x + threadIdx.x;
    int maximum = 0;
    int step = blockDim.x*gridDim.x;

    //X
    if(sliceDim == 0){
        maximum = comms.xSliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = sliceIdx;
            y = i / dims.z;
            z = i % dims.z;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            data[idx].rank = packedBuffer[i].rank;
        }
    }

    //Y
    if(sliceDim == 1){
        maximum = comms.ySliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = i % dims.x;
            y = sliceIdx;
            z = i / dims.x;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            data[idx].rank = packedBuffer[i].rank;
        }
    }

    //Z
    if(sliceDim == 2){
        maximum = comms.zSliceSize;
        for(int i = start; i < maximum; i += step){
            int x, y, z;
            x = i / dims.y;
            y = i % dims.y;
            z = sliceIdx;
            int idx = d_to1D_tiled(x, y, z, dims, tile, tileDims);
            data[idx].rank = packedBuffer[i].rank;
        }
    }
    
}

//Communication testing
void setCommTestData(Options& opt, testData* data, Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms){
    k_setCommTestData<<<opt.blocks, opt.threadsPerBlock>>>(opt, data, dims, tile, tileDims, comms);
}

void packSliceTestData(Options& opt,
                        testData* data, testData* packedBuffer,
                        Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms,
                        int sliceDim, int sliceIdx){
    k_packSliceTestData<<<opt.blocks, opt.threadsPerBlock>>>(opt, data, packedBuffer, dims, tile, tileDims, comms, sliceDim, sliceIdx);
    hipDeviceSynchronize();
}

void unpackSliceTestData(Options& opt,
                        testData* data, testData* packedBuffer,
                        Dimensions dims, Dimensions tile, Dimensions tileDims, Comms comms,
                        int sliceDim, int sliceIdx){
    k_unpackSliceTestData<<<opt.blocks, opt.threadsPerBlock>>>(opt, data, packedBuffer, dims, tile, tileDims, comms, sliceDim, sliceIdx);
    hipDeviceSynchronize();
}